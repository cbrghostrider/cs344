#include "hip/hip_runtime.h"
//Udacity HW 4
//Radix Sorting

#include "utils.h"
#include <thrust/host_vector.h>

/* Red Eye Removal
   ===============
   
   For this assignment we are implementing red eye removal.  This is
   accomplished by first creating a score for every pixel that tells us how
   likely it is to be a red eye pixel.  We have already done this for you - you
   are receiving the scores and need to sort them in ascending order so that we
   know which pixels to alter to remove the red eye.

   Note: ascending order == smallest to largest

   Each score is associated with a position, when you sort the scores, you must
   also move the positions accordingly.

   Implementing Parallel Radix Sort with CUDA
   ==========================================

   The basic idea is to construct a histogram on each pass of how many of each
   "digit" there are.   Then we scan this histogram so that we know where to put
   the output of each digit.  For example, the first 1 must come after all the
   0s so we have to know how many 0s there are to be able to start moving 1s
   into the correct position.

   1) Histogram of the number of occurrences of each digit
   2) Exclusive Prefix Sum of Histogram
   3) Determine relative offset of each digit
        For example [0 0 1 1 0 0 1]
                ->  [0 1 0 1 2 3 2]
   4) Combine the results of steps 2 & 3 to determine the final
      output location for each element and move it there

   LSB Radix sort is an out-of-place sort and you will need to ping-pong values
   between the input and output buffers we have provided.  Make sure the final
   sorted results end up in the output buffer!  Hint: You may need to do a copy
   at the end.

 */

// Note: This function has been validated against the host reference.
// Computes a histogram of the number of elements with 0 at each bitpos.
__global__ void countZerosByPositions(unsigned int* const d_in, int numElems, unsigned int* d_histo, unsigned int num_bits) {
    extern __shared__ unsigned int histo[];

    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= numElems) {
        return;
    }

    // Initialize the shared mem histo.
    if (tid == 0) {
        for (int i = 0; i < num_bits; i++) {
            histo[i] = 0;
        }
    }
    __syncthreads();

    // Count 0 valued bit at each bit_pos.
    for (unsigned int mask = 0x1, bitpos = 0; mask != 0; mask <<= 1, bitpos++) {
        unsigned int my_val = d_in[gid];
        int inc_by = ((my_val & mask) == 0x0) ? 1 : 0;
        atomicAdd(&histo[bitpos], inc_by);
    }
    __syncthreads();

    // Do the global histogram computation.
    if (tid == 0) {
        for (int bitpos = 0; bitpos < num_bits; bitpos++) {
            atomicAdd(&d_histo[bitpos], histo[bitpos]);
        }
    }
}

// Note: This function has been validated against the host reference.
// pred_cmp: The val to cmp.
// pred_mask: Determines which bit position value gets compared to.
__global__ void predicateKernel(unsigned int pred_cmp, unsigned int pred_mask, unsigned int *const d_in, unsigned int * d_preds, int numElems) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid < numElems) {
        d_preds[gid] = (((d_in[gid] & pred_mask) == pred_cmp) ? 1 : 0);
        if (d_preds[gid] != 0 && d_preds[gid] != 1) {
            printf("Strange pred value!\n"); // FIXME
        }
    }
}

// Note: Exclusive scan implementation has been verified against the host implementation!
//       This includes the local + global prefix_scan + add_scalar at the end.
// 
// Performs an exclusive prefix sum only at the block level.
// d_scan: The main input to be scanned in-place.
// d_interim: The last value for each block's prefix_sum. This will be prefix summed later outside this function.
__global__ void exclusivePrefixSum(unsigned int *const d_scan, int numElems, unsigned int* const d_interim) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= numElems) {
        return;
    }
    
    // First perform the inclusive prefix sum.
    for (int step_size = 1; step_size < blockDim.x; step_size <<= 1) {
        if (blockIdx.x == 0 && threadIdx.x == 0) {
            // printf("step_size=%d \n", step_size); // FIXME
        }
        unsigned int val_lhs = 0, val_rhs = 0;
        unsigned int block_end = (blockIdx.x + 1) * blockDim.x;
        if (gid + step_size < block_end) { 
            val_lhs = d_scan[gid];
            val_rhs = d_scan[gid + step_size]; 
        }
        __syncthreads();
        if (gid + step_size < block_end) { 
            d_scan[gid + step_size] = val_lhs + val_rhs; 
        }
        __syncthreads();
    }

    int tid = threadIdx.x;
    // Now write to d_interim if needed.
    if (tid == 0 && d_interim != nullptr) {
        int read_index = gid + blockDim.x - 1;
        read_index = min(read_index, numElems - 1);  // clamp for the last block!
        int write_index = blockIdx.x;
        d_interim[write_index] = d_scan[read_index];
        // if (d_interim[write_index]) printf("d_interim[%d] = %d, numElems=%d\n", blockIdx.x, d_interim[write_index], numElems);  // FIXME
    }
    __syncthreads();

    // Now make it an exclusive prefix sum.
    unsigned int val = 0;    
    if (tid != 0) {
        val = d_scan[gid-1];
    }
    __syncthreads();
    d_scan[gid] = val;
    if (d_scan[gid] >= numElems && nullptr != d_interim) {
        printf("[t: %d; b: %d] Unknown scan value: %u, numElems = %d\n", threadIdx.x, blockIdx.x, d_scan[gid], numElems);
    }
    __syncthreads();
    
}

// Takes the i-th indexed value from d_interim, and adds it to the i-th block elements.
__global__ void addScalar(unsigned int* const d_scan, int numElems, unsigned int* const d_interim) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= numElems) {
        return;
    }
    d_scan[gid] = d_scan[gid] + d_interim[blockIdx.x];
}

// d_histo: The histogram of count of zeroes; bitpos wise.
// bitpos: The bit position we are dealing with in this iteration.
// d_in_val and d_in_pos are the inputs.
// d_out_val and d_out_pos are the outputs.
// If d_preds at index is true, then it scatters the element at that location to the location indicated in d_scan's index.
// d_histo, bitpos, and use_offset are used to offset the location (needed for 1's).
// Both val and pos are moved identically.
__global__ void scatterWithOffsetIfPred(const unsigned int * const d_histo, int bitpos, bool use_offset,
                                        const unsigned int* const d_preds, const unsigned int* const d_scan, 
                                        const unsigned int* const d_in_vals, const unsigned int* const d_in_pos,
                                        unsigned int* const d_out_vals, unsigned int* const d_out_pos,
                                        int numElems) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= numElems) {
        return;
    }
    if (d_preds[gid] == 0) {
        return;
    }
    if (d_preds[gid] != 1) { // FIXME
        printf("[t: %d; b: %d] Unknown pred value: %u\n", threadIdx.x, blockIdx.x, d_preds[gid]);
    }
    if (d_scan[gid] >= numElems) {  // FIXME
        // printf("[t: %d; b: %d] Unknown scan value: %u\n", threadIdx.x, blockIdx.x, d_scan[gid]);
    }
    unsigned int move_to_index = d_scan[gid];
    if (use_offset) {
        move_to_index += d_histo[bitpos];
    }
    if (move_to_index < numElems) {
        // FIXME: Why is d_scan[gid] negative sometimes?
        // printf("[t: %d; b: %d] move_to_index = %u (%u + %u offset) (use_offset=%d), numElems = %d\n", threadIdx.x, blockIdx.x, move_to_index, d_scan[gid], d_histo[bitpos], use_offset, numElems);  // FIXME
        d_out_vals[move_to_index] = d_in_vals[gid];
        d_out_pos[move_to_index] = d_in_pos[gid];
    }
    else {
        // printf("[t: %d; b: %d] move_to_index = %u (%u + %u offset) (use_offset=%d), numElems = %d\n", threadIdx.x, blockIdx.x, move_to_index, d_scan[gid], d_histo[bitpos], use_offset, numElems);  // FIXME
    }
    // __syncthreads(); // WHY?
}

//unsigned int h_histogram[32] = { 0 };

void compute_host_reference(unsigned int* const d_inputVals,
    unsigned int* const d_inputPos,
    unsigned int* const d_outputVals,
    unsigned int* const d_outputPos,
    const size_t numElems) {

    // Count how many are not in order.
    /*unsigned int* h_sorted1 = static_cast<unsigned int*>(malloc(sizeof(unsigned int) * numElems));
    checkCudaErrors(hipMemcpy(h_sorted1, d_inputVals, sizeof(unsigned int) * numElems, hipMemcpyDeviceToHost));
    int count = 0;
    for (int i = 1; i < numElems; i++) {
        if (h_sorted1[i - 1] > h_sorted1[i]) {
            count++;
        }
    }
    printf("Total unsorted on device before: %d\n", count); */

    //for (int i = 0; i < numElems; i++) {
    //    unsigned int val = h_sorted1[i];
    //    for (unsigned int mask = 0x1, bitpos=0; mask != 0; mask <<=1, bitpos++) {
    //        if ((val & mask) == 0x0) {
    //            h_histogram[bitpos]++;
    //        }
    //    }        
    //}
    //printf("Host   histo: \n");
    //for (unsigned int mask = 0x1, bitpos = 0; mask != 0; mask <<= 1, bitpos++) {
    //    printf("[%d] = %d; \n", bitpos, h_histogram[bitpos]);
    //}    
    //printf("\n");
    // 
    // free(h_sorted1);
}

void test_exclusiveScan() {

    // Test the exclusive prefix scan.
    const int NUM_BLOCKS = 1024;
    const int NUM_THREADS = 1024;
    const int NUM_ELEMENTS = NUM_BLOCKS * NUM_THREADS;
    unsigned int* h_test_pfsc = static_cast<unsigned int*>(malloc(sizeof(unsigned int*) * NUM_ELEMENTS));
    for (int i = 0; i < NUM_ELEMENTS; i++) {
        h_test_pfsc[i] = 1;
    }
    unsigned int* d_test_pfsc, *d_interim;
    checkCudaErrors(hipMalloc(&d_test_pfsc, sizeof(unsigned int) * NUM_ELEMENTS));
    checkCudaErrors(hipMalloc(&d_interim, sizeof(unsigned int) * NUM_BLOCKS));
    checkCudaErrors(hipMemcpy(d_test_pfsc, h_test_pfsc, sizeof(unsigned int) * NUM_ELEMENTS, hipMemcpyHostToDevice));

    exclusivePrefixSum << <NUM_BLOCKS, NUM_THREADS >> > (d_test_pfsc, NUM_ELEMENTS, d_interim);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    exclusivePrefixSum << <1, NUM_BLOCKS >> > (d_interim, NUM_BLOCKS, nullptr);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    addScalar << <NUM_BLOCKS, NUM_THREADS >> > (d_test_pfsc, NUM_ELEMENTS, d_interim);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    checkCudaErrors(hipMemcpy(h_test_pfsc, d_test_pfsc, sizeof(unsigned int) * NUM_ELEMENTS, hipMemcpyDeviceToHost));
    bool failed = false;
    for (int i = 0; i < NUM_ELEMENTS; i++) {
        if (h_test_pfsc[i] != i) {
            printf("FAILED!! exclusive scan at index: %d; want=%d, got=%d\n", i, i, h_test_pfsc[i]);
            failed = true;
            break;
        }
    }
    if (!failed) {
        printf("Exclusive scan implementation on host vs. device matches for 1M elements!\n");
    }
}

void test_predicateKernel() {
    const int NUM_THREADS = 1024;
    const int NUM_BLOCKS = 1024;
    const int NUM_VALUES = NUM_THREADS * NUM_BLOCKS;
    unsigned int* values = static_cast<unsigned int*>(malloc(sizeof(unsigned int) * NUM_VALUES));
    unsigned int* h_out = static_cast<unsigned int*>(malloc(sizeof(unsigned int) * NUM_VALUES));
    for (int i = 0; i < NUM_VALUES; i++) {
        values[i] = i;
    }
    unsigned int* d_values, * d_out;
    checkCudaErrors(hipMalloc(&d_out, sizeof(unsigned int) * NUM_VALUES));
    checkCudaErrors(hipMalloc(&d_values, sizeof(unsigned int) * NUM_VALUES));
    checkCudaErrors(hipMemcpy(d_values, values, sizeof(unsigned int)* NUM_VALUES, hipMemcpyHostToDevice));

    // Test bitpos 0, value 0.
    bool failed = false;
    predicateKernel << <NUM_BLOCKS, NUM_THREADS >> > (0x0, 0x1, d_values, d_out, NUM_VALUES);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipMemcpy(h_out, d_out, sizeof(unsigned int) * NUM_VALUES, hipMemcpyDeviceToHost));
    // Verify.    
    for (int i = 0; i < NUM_VALUES; i++) {
        if (i % 2 == 0 && h_out[i] != 1) {
            printf("Mismatch case 1.1! ");
            failed = true;
            break;
        }
        else if (i % 2 == 1 && h_out[i] != 0) {
            printf("Mismatch case 1.2! ");
            failed = true;
            break;
        }
    }

    // Test bitpos 0, value 1.
    predicateKernel << <NUM_BLOCKS, NUM_THREADS >> > (0x1, 0x1, d_values, d_out, NUM_VALUES);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipMemcpy(h_out, d_out, sizeof(unsigned int) * NUM_VALUES, hipMemcpyDeviceToHost));
    // Verify.    
    for (int i = 0; i < NUM_VALUES; i++) {
        if (i % 2 == 0 && h_out[i] != 0) {
            printf("Mismatch case 2.1! ");
            failed = true;
            break;
        }
        else if (i % 2 == 1 && h_out[i] != 1) {
            printf("Mismatch case 2.2! ");
            failed = true;
            break;
        }
    }
    // Test bitpos 1, value 0.
    predicateKernel << <NUM_BLOCKS, NUM_THREADS >> > (0x0, 0x2, d_values, d_out, NUM_VALUES);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipMemcpy(h_out, d_out, sizeof(unsigned int) * NUM_VALUES, hipMemcpyDeviceToHost));
    // Verify.    
    for (int i = 0; i < NUM_VALUES; i++) {
        if ((i / 2) % 2 == 0 && h_out[i] != 1) {
            printf("Mismatch case 3.1! ");
            failed = true;
            break;
        }
        else if ((i / 2) % 2 == 1 && h_out[i] != 0) {
            printf("Mismatch case 3.2! ");
            failed = true;
            break;
        }
    }

    // Test bitpos 1, value 1.
    predicateKernel << <NUM_BLOCKS, NUM_THREADS >> > (0x2, 0x2, d_values, d_out, NUM_VALUES);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipMemcpy(h_out, d_out, sizeof(unsigned int) * NUM_VALUES, hipMemcpyDeviceToHost));
    // Verify.    
    for (int i = 0; i < NUM_VALUES; i++) {
        if ((i/2) % 2 == 0 && h_out[i] != 0) {
            printf("Mismatch case 4.1! ");
            failed = true;
            break;
        }
        else if ((i/2) % 2 == 1 && h_out[i] != 1) {
            printf("Mismatch case 4.2! ");
            failed = true;
            break;
        }
    }

    printf("Collective predicate kernel tests: %s!\n", (failed ? "FAILED" : "PASSED"));

    checkCudaErrors(hipFree(d_out));
    checkCudaErrors(hipFree(d_values));
    free(values);
    free(h_out);
}

void your_sort(unsigned int* const d_inputVals,
               unsigned int* const d_inputPos,
               unsigned int* const d_outputVals,
               unsigned int* const d_outputPos,
               const size_t numElems)
{ 
 
    test_exclusiveScan();
    test_predicateKernel();
    // return;  // FIXME

    const int NUM_THREADS = 1024;    
    const int NUM_BLOCKS = numElems / NUM_THREADS + 1;
    dim3 block(NUM_THREADS);
    dim3 grid(NUM_BLOCKS);

    compute_host_reference(d_inputVals, d_inputPos, d_outputVals, d_outputPos, numElems);

    // Histogram counts the number of zeros in each bit position.
    unsigned int num_bits = sizeof(unsigned int) * 8;
    unsigned int* d_histo;
    checkCudaErrors(hipMalloc(&d_histo, sizeof(unsigned int) * num_bits));
    checkCudaErrors(hipMemset(d_histo, 0, sizeof(unsigned int) * num_bits));
    countZerosByPositions << <block, grid, sizeof(unsigned int) * num_bits >> > (d_inputVals, numElems, d_histo, num_bits);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    //unsigned int* h_histo = static_cast<unsigned int *>(malloc(sizeof(unsigned int) * num_bits));
    //checkCudaErrors(hipMemcpy(h_histo, d_histo, sizeof(unsigned int) * num_bits, hipMemcpyDeviceToHost));
    //printf("Device: histo [ \n");
    //for (int i = 0; i < num_bits; i++) {
    //    printf("[%d] = %d; \n", i, h_histo[i]);
    //}
    //printf("\n");

    unsigned int* d_preds, *d_scan, *d_interim;
    checkCudaErrors(hipMalloc(&d_preds, sizeof(unsigned int) * numElems));
    checkCudaErrors(hipMalloc(&d_scan, sizeof(unsigned int) * numElems));    
    checkCudaErrors(hipMalloc(&d_interim, sizeof(unsigned int) * NUM_BLOCKS));

    // swapped indicates whether the input and output are temporarily swapped.
    bool swapped = false;  
    unsigned int* d_pi_val = d_inputVals;
    unsigned int* d_pi_pos = d_inputPos;
    unsigned int* d_po_val = d_outputVals;
    unsigned int* d_po_pos = d_outputPos;

    for (unsigned int mask = 0x1, bitpos=0; mask != 0; mask <<= 1, bitpos++) {
        // Perform predicate operations for value 0 at bitpos.
        predicateKernel << <grid, block >> > (0x0, mask, d_pi_val, d_preds, numElems);
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

        // Perform scan: Local prefix sum + global prefix sum + block-wide adds.
        checkCudaErrors(hipMemcpy(d_scan, d_preds, sizeof(unsigned int) * numElems, hipMemcpyDeviceToDevice));
        exclusivePrefixSum << <grid, block>> > (d_scan, numElems, d_interim);
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
        exclusivePrefixSum << <1, NUM_BLOCKS>> > (d_interim, NUM_BLOCKS, nullptr);
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());        
        addScalar << <grid, block>> > (d_scan, numElems, d_interim);
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

        // Perform scatter of elements.
        scatterWithOffsetIfPred << <grid, block>> > (d_histo, bitpos, false, d_preds, d_scan, d_pi_val, d_pi_pos, d_po_val, d_po_pos, numElems);
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

        // Perform predicate operations for value 1 at bitpos.
        predicateKernel << <grid, block >> > (mask, mask, d_pi_val, d_preds, numElems);
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

        // Perform scan: Local prefix sum + global prefix sum + block-wide adds.
        checkCudaErrors(hipMemcpy(d_scan, d_preds, sizeof(unsigned int) * numElems, hipMemcpyDeviceToDevice));
        exclusivePrefixSum << <grid, block >> > (d_scan, numElems, d_interim);
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
        exclusivePrefixSum << <1, NUM_BLOCKS >> > (d_interim, NUM_BLOCKS, nullptr);
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
        addScalar << <grid, block >> > (d_scan, numElems, d_interim);
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

        // Perform scatter of elements.
        scatterWithOffsetIfPred << <grid, block >> > (d_histo, bitpos, true, d_preds, d_scan, d_pi_val, d_pi_pos, d_po_val, d_po_pos, numElems);
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

        // flip the swap.
        swapped = !swapped;
        if (swapped) {
            d_pi_val = d_outputVals;
            d_pi_pos = d_outputPos;
            d_po_val = d_inputVals;
            d_po_pos = d_inputPos;
        } else {
            d_pi_val = d_inputVals;
            d_pi_pos = d_inputPos;
            d_po_val = d_outputVals;
            d_po_pos = d_outputPos;
        }
    }
    if (!swapped) {  // check for negative since it is flipped at the end of the last iteration at exit!
        checkCudaErrors(hipMemcpy(d_outputVals, d_inputVals, sizeof(unsigned int) * numElems, hipMemcpyDeviceToDevice));
        checkCudaErrors(hipMemcpy(d_outputPos, d_inputPos, sizeof(unsigned int) * numElems, hipMemcpyDeviceToDevice));
    }

    checkCudaErrors(hipFree(d_preds));
    checkCudaErrors(hipFree(d_scan));
    checkCudaErrors(hipFree(d_interim));
    checkCudaErrors(hipFree(d_histo));

    // Check if sorted.
    unsigned int* h_sorted = static_cast<unsigned int*>(malloc(sizeof(unsigned int) * numElems));
    checkCudaErrors(hipMemcpy(h_sorted, d_outputVals, sizeof(unsigned int) * numElems, hipMemcpyDeviceToHost));
    int count = 0;
    for (int i = 1; i < numElems; i++) {
        if (h_sorted[i-1] > h_sorted[i]) {
            // printf("Device output was not sorted at indices [%d]=%d and [%d]=%d\n", i-1, h_sorted[i-1], i, h_sorted[i]);
            count++;
            // break;
        }
    }
    free(h_sorted);
    printf("Total unsorted on device: %d\n", count);
}
